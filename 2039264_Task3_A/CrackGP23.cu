#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define RAW_PASSWORD_SIZE  11
#define DECRYPTED_PASSWORD_SIZE (4 + 1)

#define CRYPT_TEST_COUNT 1


// compile with nvcc CrackGP23.cu -o crackGP23
/**************************
compile with

	 cc nvcc nvcc CrackGP23.cu -o crackGP23
	 
***************************/	
 

// the cuda crypt function takes in a raw password of ONLY 2 letters (ONLY LOWERCASE) and 2 numbers.
// Your task is to take this function and create a "__device__ crypt" function which can be used within the "__global__"
// function. That way, you can encrypt all combinations and check this password with rawPassword to
// determine whether a match has been found.

__device__ void cuda_device(unsigned char* rawPassword, unsigned char* newPassword)
{
	newPassword[0] = rawPassword[0] + 2;
	newPassword[1] = rawPassword[0] - 2;
	newPassword[2] = rawPassword[0] + 1;
	newPassword[3] = rawPassword[1] + 3;
	newPassword[4] = rawPassword[1] - 3;
	newPassword[5] = rawPassword[1] - 1;
	newPassword[6] = rawPassword[2] + 2;
	newPassword[7] = rawPassword[2] - 2;
	newPassword[8] = rawPassword[3] + 4;
	newPassword[9] = rawPassword[3] - 4;
	newPassword[10] = '\0';

	for(int i =0; i<10; i++){
		if(i >= 0 && i < 6){ //checking all lower case letter limits
			if(newPassword[i] > 122){
				newPassword[i] = (newPassword[i] - 122) + 97;
			}else if(newPassword[i] < 97){
				newPassword[i] = (97 - newPassword[i]) + 97;
			}
		}else{ //checking number section
			if(newPassword[i] > 57){
				newPassword[i] = (newPassword[i] - 57) + 48;
			}else if(newPassword[i] < 48){
				newPassword[i] = (48 - newPassword[i]) + 48;
			}
		}
	}
}

__global__ void cuda_Crypt(unsigned char* raw_password, unsigned char* decrypted_password)
{
	int block_idx = blockIdx.x;
	int thread_idx = threadIdx.x;

	unsigned char current_password[DECRYPTED_PASSWORD_SIZE];
	unsigned char encrypted_password[RAW_PASSWORD_SIZE];

	for(int i = 0; i < 10; i++)
	{
		for(int j = 0; j < 10; j++)
		{
			char is_correct = 0;
			current_password[0] = 97 + block_idx;
			current_password[1] = 97 + thread_idx;
			current_password[2] = 48 + i;
			current_password[3] = 48 + j;
			cuda_device(current_password, encrypted_password);

			for(int k = 0; k < RAW_PASSWORD_SIZE; k++)
			{
				if(encrypted_password[k] == raw_password[k])
				{
					++is_correct;
				}
			}

			if(is_correct == RAW_PASSWORD_SIZE)
			{
				for(int k = 0; k < DECRYPTED_PASSWORD_SIZE; k++)
				{
					decrypted_password[k] = current_password[k];
				}
			}
		}
	}
}

void crypt()
{

	unsigned char rawPassword[RAW_PASSWORD_SIZE] = "y}zous4071";
	unsigned char decryptedPassword[DECRYPTED_PASSWORD_SIZE] = {0};

	const int blocks = 26;
	const int threads = 26;

	// declare GPU memory pointers
	unsigned char * d_in;
	unsigned char * d_out;

	// allocate GPU memory
	hipMalloc((void**) &d_in, RAW_PASSWORD_SIZE);
	hipMalloc((void**) &d_out, DECRYPTED_PASSWORD_SIZE);

	hipMemcpy(d_in, rawPassword, RAW_PASSWORD_SIZE, hipMemcpyHostToDevice);

	// launch the kernel
	cuda_Crypt<<<blocks, threads>>>(d_in, d_out);

	// copy back the result array to the CPU
	hipMemcpy(decryptedPassword, d_out, DECRYPTED_PASSWORD_SIZE, hipMemcpyDeviceToHost);

	//free(image);
	//free(host_imageInput);
	hipFree(d_in);
	hipFree(d_out);

	decryptedPassword[DECRYPTED_PASSWORD_SIZE - 1] = '\0';
}

int main(int argc, char *argv[]){
// y}zous4071 GP23
	printf("CUDA - Password Cracking of 2 Upper Case Letter And 2 Integer Numbers  \n");
	printf("No Of Loops : %d \n", CRYPT_TEST_COUNT);

	struct timespec start, finish;
	double test_time[CRYPT_TEST_COUNT];
	double total_time = 0, total_square_time = 0, average_time = 0, variance_time = 0;

	for(int i = 0; i < CRYPT_TEST_COUNT; i++)
	{
		clock_gettime(CLOCK_REALTIME, &start);
		crypt();
		clock_gettime(CLOCK_REALTIME, &finish);

		long seconds = finish.tv_sec - start.tv_sec;
	    long ns = finish.tv_nsec - start.tv_nsec;

	    if (start.tv_nsec > finish.tv_nsec)
	    {
	    	--seconds;
	    	ns += 1000000000;
	    }

	    double time_elapsed = (double)seconds + (double)ns/(double)1000000000;

		test_time[i] = time_elapsed;
		total_time += time_elapsed;
		printf("%10s %10s \n", "Loop", "Time(seconds)");
		printf("%5d %15.3f \n", (i + 1), time_elapsed);
		fflush(stdout);
	}

	average_time = total_time / CRYPT_TEST_COUNT;

	for(int i = 0; i < CRYPT_TEST_COUNT; i++)
	{
		total_square_time += pow(test_time[i] - average_time, 2);
	}

	variance_time = sqrt(total_square_time / CRYPT_TEST_COUNT);
	printf("\n AVERAGE TIME %5.3f +/- %5.3f seconds \n", average_time, variance_time);
	return 0;
}
