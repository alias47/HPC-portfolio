#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>

#include "lodepng.h"

#define CRYPT_TEST_COUNT 10

/**************************
compile with

	 cc nvcc imageblur.cu lodepng.cu -o imageblur 	 
***************************/	



__global__ void blur_gaussian_GPUFunction(unsigned char * gpu_imageOuput, unsigned char * gpu_imageInput,
										  unsigned int height, unsigned int width, unsigned int filter_size){

	int r = 0;
	int g = 0;
	int b = 0;
	int t = 0;
	int sumR = 0;
	int sumG = 0;
	int sumB = 0;
	int count = 0;

	unsigned int half_filter_size = filter_size / 2;

	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int pixel = idx*4;

	t = gpu_imageInput[3 + pixel];

	for(int filter_y = blockIdx.x - half_filter_size; filter_y < blockIdx.x + half_filter_size; filter_y++)
	{
		/* Check boundary condition along y - direction ( If we have gone out of bounds) */
		if(filter_y >= 0 && filter_y < height)
		{
			for(int filter_x = threadIdx.x - half_filter_size; filter_x < threadIdx.x + half_filter_size; filter_x++)
			{
				/* Check boundary condition along x - direction ( If we have gone out of bounds) */
				if(filter_x >= 0 && filter_x < width)
				{
					r = gpu_imageInput[4 * blockDim.x * filter_y + 4 * filter_x + 0];
					g = gpu_imageInput[4 * blockDim.x * filter_y + 4 * filter_x + 1];
					b = gpu_imageInput[4 * blockDim.x * filter_y + 4 * filter_x + 2];
					sumR += r; /* Compute sum of R values to calculate average */
					sumG += g; /* Compute sum of G values to calculate average */
					sumB += b; /* Compute sum of B values to calculate average */
					++count; /* Compute no. of values to calculate average */
				}
			}
		}
	}

	gpu_imageOuput[pixel] = (sumR / count);
	gpu_imageOuput[1 + pixel] = (sumG / count);
	gpu_imageOuput[2 + pixel] = (sumB / count);
	gpu_imageOuput[3 + pixel] = t;
}

void blur_gaussian(){

	unsigned int error;
	unsigned int encError;
	unsigned char* image;
	unsigned int width;
	unsigned int height;
	unsigned int filter_size = 7;

	const char* filename = "man.png";
	const char* newFileName = "generatedman.png";

	error = lodepng_decode32_file(&image, &width, &height, filename);
	if(error){
		printf("error %u: %s\n", error, lodepng_error_text(error));
	}

	const int ARRAY_SIZE = width*height*4;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned char);

	unsigned char host_imageInput[ARRAY_SIZE * 4];
	unsigned char host_imageOutput[ARRAY_SIZE * 4];

	for (int i = 0; i < ARRAY_SIZE; i++) {
		host_imageInput[i] = image[i];
	}

	// GPU memory pointers
	unsigned char * d_in;
	unsigned char * d_out;
	unsigned int* gpu_height;
	unsigned int* gpu_width;
	unsigned int* gpu_filter_size;

	// allocatation of GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES);
	hipMalloc((void**) &d_out, ARRAY_BYTES);
	hipMalloc((void**) &gpu_height, sizeof(int));
	hipMalloc((void**) &gpu_width, sizeof(int));
	hipMalloc((void**) &gpu_filter_size, sizeof(int));

	hipMemcpy(d_in, host_imageInput, ARRAY_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(gpu_height, &height, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(gpu_width, &width, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(gpu_filter_size, &filter_size, sizeof(int), hipMemcpyHostToDevice);

	// Launch of Kernal
	blur_gaussian_GPUFunction<<<height, width>>>(d_out, d_in, height, width, filter_size);

	
	hipMemcpy(host_imageOutput, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	encError = lodepng_encode32_file(newFileName, host_imageOutput, width, height);
	if(encError){
		printf("Error %u: %s\n", error, lodepng_error_text(encError));
	}

	//free(image);
	//free(host_imageInput);
	hipFree(d_in);
	hipFree(d_out);
}

int main(int argc, char *argv[]){

	printf("Blur Image\n");
	printf("No Of Loops : %d \n", CRYPT_TEST_COUNT);

	struct timespec start, finish;
	double test_time[CRYPT_TEST_COUNT];
	double total_time = 0, total_square_time = 0, average_time = 0, variance_time = 0;

	for(int i = 0; i < CRYPT_TEST_COUNT; i++)
	{
		clock_gettime(CLOCK_REALTIME, &start);
		blur_gaussian();
		clock_gettime(CLOCK_REALTIME, &finish);

		long seconds = finish.tv_sec - start.tv_sec;
	    long ns = finish.tv_nsec - start.tv_nsec;

	    if (start.tv_nsec > finish.tv_nsec)
	    {
	    	--seconds;
	    	ns += 1000000000;
	    }

	    double time_elapsed = (double)seconds + (double)ns/(double)1000000000;

		test_time[i] = time_elapsed;
		total_time += time_elapsed;
		printf("%10s %10s \n", "Loops", "Time(seconds)");
		printf("%5d %15.3f \n", (i + 1), time_elapsed);
		fflush(stdout);
	}

	average_time = total_time / CRYPT_TEST_COUNT;

	for(int i = 0; i < CRYPT_TEST_COUNT; i++)
	{
		total_square_time += pow(test_time[i] - average_time, 2);
	}

	variance_time = sqrt(total_square_time / CRYPT_TEST_COUNT);
	printf("\n Average Time %5.3f +/- %5.3f seconds \n", average_time, variance_time);
	return 0;
}
